#include <stdio.h>
#include<hip/hip_runtime.h>
#include <stdlib.h>
#include<chrono>
#include <vector>
#include <iostream>
const int X=5716,Y=3731,W=50;
// カーネル(GPUの関数)
__global__ void cudaKernel(int *gpu){
    // スレッドID

    int xid=blockIdx.x*blockDim.x+threadIdx.x;
    int yid=blockIdx.y*blockDim.y+threadIdx.y;
    //W近傍の和を愚直にとる
    int V=0,kaz=0;
    for(int dy=0;dy<W;dy++){
        for(int dx=0;dx<W;dx++){
            int sx=xid+dx;
            int sy=yid+dy;
            if(sx<0||sy<0||sx>=X||sy>=Y){continue;}
            V+=gpu[sy*X+sx];
            kaz++;
        }
    }
    __syncthreads();
    if(0<=yid&&yid+W<Y&&0<=xid&&xid+W<X){
        gpu[yid*X+xid]=V/kaz;
    }
   
}
using namespace std;
int main(int argc, char** argv){
    
    //static int pic[Y*X];
    int* pic;
    int* picgpu;
    int * gpuwa;
    
    pic=(int*)malloc(sizeof(int)*X*Y);
    //デバイスの初期化
    //CUT_DEVICE_INIT(argc, argv);

    FILE *fpin, *fpout;
    char *fin = argv[1];
    char *fout = argv[2];

    fpin = fopen(fin, "r");
    fpout = fopen(fout, "w");
    // xの初期化
    //printf("debug%d\n",__LINE__);
    //xとyをどうするかについて考える
    //bmp形式について考える
    unsigned char ch;
    for (int i = 0; i < 3; ++i) {
        while ((ch = fgetc(fpin)) != EOF) {
            if (i != 1){ fputc(ch, fpout);}
            if(i==1){printf("%c",ch);}
            if (ch == '\n'){break;}
        }
        if (i == 1) {
            fprintf(fpout, "%d %d\n", X-W, Y-W);
            printf("%d %d\n", X-W, Y-W);
        }
    }

    for (int i = 0; i < Y; ++i) {
        for (int j = 0; j < X; ++j) {
            ch = fgetc(fpin);
            pic[i*X+j] = ch;
            //if((i*X+j)%300000==0){printf("%d\n",(int)ch);}
        }
    }
    std::cout <<"debug"<<__LINE__<<endl;
    auto startA = std::chrono::system_clock::now(); 
    //printf("debug%d\n",__LINE__);
    // デバイス(GPU)のメモリ領域確保
    hipMalloc((void**)&picgpu, sizeof(int)*X*Y);
    //cudaMalloc((void**)&gpuwa, sizeof(int)*X*Y);
    std::cout <<"debug"<<__LINE__<<endl;
    // ホスト(CPU)からデバイス(GPU)へ転送
    auto startB = std::chrono::system_clock::now(); 
    hipMemcpy(picgpu, pic, sizeof(int)*X*Y, hipMemcpyHostToDevice);
          // 計測終了時刻を保存
    //printf("debug%d\n",__LINE__);
    // スレッド数、ブロック数の設定(説明は他のページ)
    dim3 blocks((X+15)/16,(Y+15)/16);
    dim3 threads(16,16);
    auto start = std::chrono::system_clock::now(); 
    // カーネル(GPUの関数)実行
    cudaKernel<<< blocks, threads >>>(picgpu);
    auto end = std::chrono::system_clock::now(); 
    // デバイス(GPU)からホスト(CPU)へ転送
    hipMemcpy(pic, picgpu, sizeof(int)*X*Y, hipMemcpyDeviceToHost);
    //printf("debug%d\n",__LINE__);
    auto endA = std::chrono::system_clock::now(); 
    std::cout <<"debug"<<__LINE__<<endl;
    for (int i = 0; i+W < Y; ++i) {
        for (int j = 0; j+W < X; ++j) {
            fputc(pic[i*X+j], fpout);
            //if((i*X+j)%300000==0){printf("%d\n",(int)pic[i*X+j]);}
        }
    }

    //printf("debug%d\n",__LINE__);

    auto msec = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // 要した時間をミリ秒（1/1000秒）に変換して表示
    std::cout <<"keisan="<< msec << " milli sec \n";
    auto Amsec = std::chrono::duration_cast<std::chrono::milliseconds>(endA-startA).count();
    std::cout <<"All="<< Amsec << " milli sec \n";
    auto Bmsec = std::chrono::duration_cast<std::chrono::milliseconds>(endA-startB).count();
    std::cout <<"notnalloc="<< Bmsec << " milli sec \n";
    fclose(fpin);
    fclose(fpout);
    // ホストメモリ解放
    free(pic);
    
    // デバイスメモリ解放
    hipFree(picgpu);
    //cudaFree(gpuwa);
    // 終了処理
    //CUT_EXIT(argc, argv);
    return 0;
}