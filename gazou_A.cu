#include <stdio.h>
#include<hip/hip_runtime.h>

#define N 100

const int X=5716,Y=3731;
// カーネル(GPUの関数)
__global__ void cudaKernel(int gpu[]){
    // スレッドID

    int xid=blockIdx.x*blockDim.x+threadIdx.x;
    int yid=blockIdx.y*blockDim.y+threadIdx.y;
    //50近傍の和を愚直にとる
    int V=0,kaz=0;
    for(int dy=-50;dy<=50;dy++){
        for(int dx=-50;dx<=50;dy++){
            int sx=xid-dx;
            int sy=yid-dy;
            if(sx<0||sy<0||sx>X||sy>Y){continue;}
            V+=gpu[sy*X+sx];
            kaz++;
        }
    }
    __syncthreads();
    if(0<=yid&&yid<Y&&0<=xid&&xid<X){
        gpu[yid*X+xid]=V/kaz;
    }
   
}

int main(int argc, char** argv){
    
    static int pic[Y][X];
    int* picgpu;
    unsigned int * gpuwa;

    //デバイスの初期化
    //CUT_DEVICE_INIT(argc, argv);

    FILE *fpin, *fpout;
    char *fin = argv[1];
    char *fout = argv[2];

    fpin = fopen(fin, "r");
    fpout = fopen(fout, "w");
    // xの初期化
    printf("debug%d\n",__LINE__);
    //xとyをどうするかについて考える
    //bmp形式について考える
    char ch;
    for (int i = 0; i < 3; ++i) {
        while ((ch = fgetc(fpin)) != EOF) {
            if (i != 1)
                fputc(ch, fpout);
            if (ch == '\n')
                break;
        }
        if (i == 1) {
            fprintf(fpout, "%d %d\n", Y, X);
        }
    }

    for (int i = 0; i < Y; ++i) {
        for (int j = 0; j < X; ++j) {
            ch = fgetc(fpin);
            pic[i][j] = ch;
        }
    }
    printf("debug%d\n",__LINE__);
    // デバイス(GPU)のメモリ領域確保
    hipMalloc((void**)&picgpu, sizeof(int)*X*Y);
    hipMalloc((void**)&gpuwa, sizeof(int)*X*Y);

    // ホスト(CPU)からデバイス(GPU)へ転送
    hipMemcpy(pic, picgpu, sizeof(int)*X*Y, hipMemcpyHostToDevice);

    printf("debug%d\n",__LINE__);
    // スレッド数、ブロック数の設定(説明は他のページ)
    dim3 blocks(16,16);
    dim3 threads((Y+15)/16,(X+15)/16);

    // カーネル(GPUの関数)実行
    //cudaKernel<<< blocks, threads >>>(picgpu);

    // デバイス(GPU)からホスト(CPU)へ転送
    hipMemcpy(pic, picgpu, sizeof(int)*X*Y, hipMemcpyDeviceToHost);
    printf("debug%d\n",__LINE__);
    
    for (int i = 0; i < Y; ++i) {
        for (int j = 0; j < X; ++j) {
            fputc(pic[i][j], fpout);
        }
    }
    printf("debug%d\n",__LINE__);
    fclose(fpin);
    fclose(fpout);
    // ホストメモリ解放
    //free(pic);
    
    // デバイスメモリ解放
    hipFree(picgpu);
    hipFree(gpuwa);
    // 終了処理
    //CUT_EXIT(argc, argv);
    return 0;
}