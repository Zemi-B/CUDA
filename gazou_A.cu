#include "hip/hip_runtime.h"
#include <stdio.h>


#define N 100

const int xsize=5716,ysize=3731;
// カーネル(GPUの関数)
__global__ void cudaKernel(int *gpu){
    // スレッドID

    int xid=blockIdx.x*blockDim.x+threadIdx.x;
    int yid=blockIdx.y*blockDim.y+threadIdx.y;
    //50近傍の和を愚直にとる
    int V=0,kaz=0;
    for(dy=-50;dy<=50;dy++){
        for(dx=-50;dx<=50;dy++){
            int sx=xid-dx;
            int sy=yid-dy;
            if(sx<0||sy<0||sx>xsize||sy>ysize){continue;}
            V+=gpu[sy][sx];
            kaz++;
        }
    }
    __syncthreads();
    gpu[yid][xid]=V/kaz;
}

int main(int argc, char** argv){
    
    int pic[ysize][xsize];
    int* picgpu;
    unsigned int * gpuwa;

    //デバイスの初期化
    CUT_DEVICE_INIT(argc, argv);

    // xの初期化

    //xとyをどうするかについて考える
    //bmp形式について考える
    char ch;
    for (int i = 0; i < 3; ++i) {
        while ((ch = fgetc(fpin)) != EOF) {
            if (i != 1)
                fputc(ch, fpout);
            if (ch == '\n')
                break;
        }
        if (i == 1) {
            fprintf(fpout, "%d %d\n", Y - W, X - W);
        }
    }

    for (int i = 0; i < Y; ++i) {
        for (int j = 0; j < X; ++j) {
            ch = fgetc(fpin);
            pic[i][j] = ch;
        }
    }

    // デバイス(GPU)のメモリ領域確保
    CUDA_SAFE_CALL(hipMalloc((void**)&picgpu, sizeof(int)*X*Y));
    CUDA_SAFE_CALL(hipMalloc((void**)&gpuwa, sizeof(int)*X*Y));

    // ホスト(CPU)からデバイス(GPU)へ転送
    CUDA_SAFE_CALL(hipMemcpy(pic, picgpu, sizeof(int)*X*Y, hipMemcpyHostToDevice));

    
    // スレッド数、ブロック数の設定(説明は他のページ)
    dim3 blocks(16,16);
    dim3 threads((ysize+15)/16,(xsize+15)/16);

    // カーネル(GPUの関数)実行
    cudaKernel<<< blocks, threads >>>(gpu);

    // デバイス(GPU)からホスト(CPU)へ転送
    CUDA_SAFE_CALL(hipMemcpy(pic, picgpu, sizeof(int)*X*Y, hipMemcpyDeviceToHost));

    

    // ホストメモリ解放
    free(pic);
    
    // デバイスメモリ解放
    CUDA_SAFE_CALL(hipFree(picgpu));
    CUDA_SAFE_CALL(hipFree(gpuwa));
    // 終了処理
    CUT_EXIT(argc, argv);
    return 0;
}