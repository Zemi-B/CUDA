#include <stdio.h>
#include<hip/hip_runtime.h>
#include <stdlib.h>

const int X=5716,Y=3731,W=100;
// カーネル(GPUの関数)
__global__ void cudaKernel(int *gpu){
    // スレッドID

    int xid=blockIdx.x*blockDim.x+threadIdx.x;
    int yid=blockIdx.y*blockDim.y+threadIdx.y;
    //W近傍の和を愚直にとる
    int V=0,kaz=0;
    for(int dy=0;dy<W;dy++){
        for(int dx=0;dx<W;dx++){
            int sx=xid+dx;
            int sy=yid+dy;
            if(sx<0||sy<0||sx>=X||sy>=Y){continue;}
            V+=gpu[sy*X+sx];
            kaz++;
        }
    }
    __syncthreads();
    if(0<=yid&&yid+W<Y&&0<=xid&&xid+W<X){
        gpu[yid*X+xid]=V/(kaz*2);
    }
   
}

int main(int argc, char** argv){
    
    //static int pic[Y*X];
    int* pic;
    int* picgpu;
    int * gpuwa;
    
    pic=(int*)malloc(sizeof(int)*X*Y);
    //デバイスの初期化
    //CUT_DEVICE_INIT(argc, argv);

    FILE *fpin, *fpout;
    char *fin = argv[1];
    char *fout = argv[2];

    fpin = fopen(fin, "r");
    fpout = fopen(fout, "w");
    // xの初期化
    printf("debug%d\n",__LINE__);
    //xとyをどうするかについて考える
    //bmp形式について考える
    unsigned char ch;
    for (int i = 0; i < 3; ++i) {
        while ((ch = fgetc(fpin)) != EOF) {
            if (i != 1){ fputc(ch, fpout);}
            if(i==1){printf("%c",ch);}
            if (ch == '\n'){break;}
        }
        if (i == 1) {
            fprintf(fpout, "%d %d\n", X-W, Y-W);
            printf("%d %d\n", X-W, Y-W);
        }
    }

    for (int i = 0; i < Y; ++i) {
        for (int j = 0; j < X; ++j) {
            ch = fgetc(fpin);
            pic[i*X+j] = ch;
            if((i*X+j)%300000==0){printf("%d\n",(int)ch);}
        }
    }
    printf("debug%d\n",__LINE__);
    // デバイス(GPU)のメモリ領域確保
    hipMalloc((void**)&picgpu, sizeof(int)*X*Y);
    hipMalloc((void**)&gpuwa, sizeof(int)*X*Y);

    // ホスト(CPU)からデバイス(GPU)へ転送
    hipMemcpy(picgpu, pic, sizeof(int)*X*Y, hipMemcpyHostToDevice);

    printf("debug%d\n",__LINE__);
    // スレッド数、ブロック数の設定(説明は他のページ)
    dim3 blocks(16,16);
    dim3 threads((X+15)/16,(Y+15)/16);

    // カーネル(GPUの関数)実行
    cudaKernel<<< blocks, threads >>>(picgpu);

    // デバイス(GPU)からホスト(CPU)へ転送
    hipMemcpy(pic, picgpu, sizeof(int)*X*Y, hipMemcpyDeviceToHost);
    printf("debug%d\n",__LINE__);
    
    for (int i = 0; i+W < Y; ++i) {
        for (int j = 0; j+W < X; ++j) {
            fputc(pic[i*X+j], fpout);
            if((i*X+j)%300000==0){printf("%d\n",(int)pic[i*X+j]);}
        }
    }
    printf("debug%d\n",__LINE__);
    fclose(fpin);
    fclose(fpout);
    // ホストメモリ解放
    free(pic);
    
    // デバイスメモリ解放
    hipFree(picgpu);
    hipFree(gpuwa);
    // 終了処理
    //CUT_EXIT(argc, argv);
    return 0;
}